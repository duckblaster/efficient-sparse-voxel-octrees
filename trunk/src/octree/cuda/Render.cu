#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2011, NVIDIA Corporation
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *      * Redistributions of source code must retain the above copyright
 *        notice, this list of conditions and the following disclaimer.
 *      * Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimer in the
 *        documentation and/or other materials provided with the distribution.
 *      * Neither the name of NVIDIA Corporation nor the
 *        names of its contributors may be used to endorse or promote products
 *        derived from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 *  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 *  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "Render.hpp"
#include "../io/OctreeRuntime.hpp"
#include "../io/AttachIO.hpp"

using namespace FW;

//------------------------------------------------------------------------
// Global variables.
//------------------------------------------------------------------------

__constant__ int4   c_input[(sizeof(RenderInput) + sizeof(int4) - 1) / sizeof(int4)];
__constant__ int4   c_blurLUT[BLUR_LUT_SIZE];
__device__ S32      g_warpCounter;

texture<U32, 1>     texIndexToPixel;
texture<U32, 1>     texIndexToPixelCoarse;
texture<F32, 1>     texFrameCoarseIn;
texture<uchar4, 1, hipReadModeNormalizedFloat> texTempFrameIn;
texture<uchar4, 1, hipReadModeNormalizedFloat> texAASamplesIn;

//------------------------------------------------------------------------
// Common helper functions.
//------------------------------------------------------------------------

__device__ inline const RenderInput& getInput(void)
{
    return *(const RenderInput*)c_input;
}

__device__ inline void updateCounter(PerfCounter counter, int amount = 1)
{
#ifdef ENABLE_PERF_COUNTERS
    int warpIdx = threadIdx.y + blockIdx.x * RCK_TRACE_BLOCK_HEIGHT;
    volatile S64* ptr = (S64*)getInput().perfCounters + (warpIdx * PerfCounter_Max + counter) * 33;
    ptr[threadIdx.x] += amount; // thread counter
    ptr[32] += amount; // warp counter
#endif
}

#ifdef ENABLE_PERF_COUNTERS
__device__ inline bool checkTransaction(int page)
{
    volatile __shared__ U32 buffer[RCK_TRACE_BLOCK_WIDTH * RCK_TRACE_BLOCK_HEIGHT];
    int fullIdx = threadIdx.x + threadIdx.y * RCK_TRACE_BLOCK_WIDTH;
    volatile U32* ptr = &buffer[fullIdx & -16];
    int idx = fullIdx & 15;

    // Clear buffer.

    for (int i = 0; i < 16; i++)
        ptr[i] = 0xFFFFFFFF;

    // Write address.

    ptr[idx] = page;

    // Check for duplicates.

    for (int i = 0; i < idx; i++)
        if (ptr[i] == page)
            return false;
    return true;
}
#endif

__device__ inline void updateCountersForGlobalAccess(int sizeLog2, S32* addr)
{
#ifdef ENABLE_PERF_COUNTERS
    updateCounter(PerfCounter_GlobalAccesses);
    updateCounter(PerfCounter_GlobalBytes, 1 << sizeLog2);
    if (checkTransaction((U32)addr >> ::min(sizeLog2 + 5, 7)))
        updateCounter(PerfCounter_GlobalTransactions);
#endif
}

__device__ inline void updateCountersForLocalAccess(int sizeLog2, int id)
{
#ifdef ENABLE_PERF_COUNTERS
    updateCounter(PerfCounter_LocalAccesses);
    updateCounter(PerfCounter_LocalBytes, 1 << sizeLog2);
    if (checkTransaction(id))
        updateCounter(PerfCounter_LocalTransactions);
#endif
}

//------------------------------------------------------------------------
// Utility routines.
//------------------------------------------------------------------------

#include "Util.inl"
#include "Raycast.inl"
#include "AttribLookup.inl"

//------------------------------------------------------------------------
// Private definitions.
//------------------------------------------------------------------------

#define BLUR_FACTOR 1.0f // Controls total amount of blurring. Larger than 1.0 causes everything to blur.

struct Aux // shared memory auxiliary storage
{
    U32*        framePtr;
#ifdef LARGE_RECONSTRUCTION_KERNEL
    U32*        aaSamplePtr;
#endif
#ifdef JITTER_LOD
    float       vSizeMultiplier;
#endif

    union
    {
        S32     fetchWorkTemp;

        Ray     ray;

        struct
        {
            U32 color;
            U32 alpha;
        }       aa;
    };
};

__constant__ float2 c_aa4table[4] =
{
    { 0.125f, 0.375f },
    { 0.375f, 0.875f },
    { 0.875f, 0.625f },
    { 0.625f, 0.125f }
};

//------------------------------------------------------------------------
// Ray generation.
//------------------------------------------------------------------------

__device__ Ray constructPrimaryRay(int ppos, int ridx, volatile Aux& aux)
{
    const RenderInput& input = getInput();
    float vsize = input.maxVoxelSize;
    int xsize = input.frameSize.x;

    // if coarse pass, make voxel large enough so that rays cannot accidentally get past it
    if (input.flags & RenderFlags_CoarsePass)
    {
        vsize = (float)input.coarseSize * 2.83f; // sqrt(8)
        xsize = input.coarseFrameSize.x;
    } else
    {
#ifdef JITTER_LOD
        // perturb randomly
        U32 a = ppos;
        U32 b = ridx;
        U32 c = 0x9e3779b9u;
        jenkinsMix(a, b, c);
        float f = (float)c / ((float)(1u << 31) * 2.f);
        f = .5f + .5f*f;
        aux.vSizeMultiplier = f;
        vsize *= f;
#endif
    }

    // find ray coordinates
    int pixely = ppos / xsize;
    int pixelx = ppos - (pixely * xsize);
    F32 fx = pixelx;
    F32 fy = pixely;

    if (input.flags & RenderFlags_CoarsePass)
    {
        fx *= (float)input.coarseSize;
        fy *= (float)input.coarseSize;
    } else
    {
        if (input.aaRays == 1)
        {
            fx += .5f; // center of pixel
            fy += .5f;
        } else if (input.aaRays == 4)
        {
            int aidx = (ridx & 3);
            fx += c_aa4table[aidx].x;
            fy += c_aa4table[aidx].y;
        }
    }

    F32 tmin = 0.f;
    if (getInput().flags & RenderFlags_UseCoarseData)
    {
        // fetch tmin
        int bx = pixelx / input.coarseSize;
        int by = pixely / input.coarseSize;
        int bidx = bx + by * input.coarseFrameSize.x;
        F32 tmin0 = tex1Dfetch(texFrameCoarseIn, bidx);
        F32 tmin1 = tex1Dfetch(texFrameCoarseIn, bidx+1);
        F32 tmin2 = tex1Dfetch(texFrameCoarseIn, bidx+input.coarseFrameSize.x);
        F32 tmin3 = tex1Dfetch(texFrameCoarseIn, bidx+input.coarseFrameSize.x+1);
        tmin = fminf(fminf(tmin0, tmin1), fminf(tmin2, tmin3));
        tmin = fminf(tmin, 0.9999f);
    }

    const Mat4f& vtc = input.octreeMatrices.viewportToCamera;
    const Mat4f& cto = input.octreeMatrices.cameraToOctree;

    float4 pos = make_float4(
        vtc.m00 * fx + vtc.m01 * fy + vtc.m03,
        vtc.m10 * fx + vtc.m11 * fy + vtc.m13,
        vtc.m20 * fx + vtc.m21 * fy + vtc.m23,
        vtc.m30 * fx + vtc.m31 * fy + vtc.m33);

    float3 near = make_float3(
        pos.x - vtc.m02,
        pos.y - vtc.m12,
        pos.z - vtc.m22);
    float near_sz = input.octreeMatrices.pixelInOctree * vsize;

    float3 diff = make_float3(
        vtc.m32 * pos.x - vtc.m02 * pos.w,
        vtc.m32 * pos.y - vtc.m12 * pos.w,
        vtc.m32 * pos.z - vtc.m22 * pos.w);
    float diff_sz = near_sz * vtc.m32;

    float a = 1.0f / (pos.w - vtc.m32);
    float b = 2.0f * a / fmaxf(pos.w + vtc.m32, 1.0e-8f);
    float c = tmin * b;

    Ray ray;
    ray.orig = near * a - diff * c;
    ray.dir  = diff * (c - b);
    ray.orig_sz = near_sz * a - diff_sz * c;
    ray.dir_sz  = diff_sz * (c - b);

    ray.orig = cto * ray.orig;
    ray.dir = make_float3(
        cto.m00 * ray.dir.x + cto.m01 * ray.dir.y + cto.m02 * ray.dir.z,
        cto.m10 * ray.dir.x + cto.m11 * ray.dir.y + cto.m12 * ray.dir.z,
        cto.m20 * ray.dir.x + cto.m21 * ray.dir.y + cto.m22 * ray.dir.z);
    return ray;
}

//------------------------------------------------------------------------
// Ray processing.
//------------------------------------------------------------------------

__device__ U32 processPrimaryRay(volatile Ray& ray, volatile F32& vSizeMultiplier)
{
    // Cast primary ray.

    CastResult castRes;
    CastStack stack;
    castRay(castRes, stack, ray);

    // Handle visualizations.

    if (getInput().flags & RenderFlags_VisualizeIterations)
    {
        F32 v = 255.0f * (F32)castRes.iter / 64.0f;
        return toABGR(make_float4(v, v, v, 0.0f));
    }
    else if (getInput().flags & RenderFlags_VisualizeRaycastLevel)
    {
        F32 v = 0.0f;
        if (castRes.t <= 1.0f)
            v = 255.0f - ((F32)CAST_STACK_DEPTH - (F32)castRes.stackPtr) * (255.0f / 18.0f);
        return toABGR(make_float4(v * 0.5f, v, v * 0.5f, 0.0f));
    }

    // Initialize light and incident vectors.

    float3 L = make_float3(0.3643f, 0.3535f, 0.8616f);
    float3 I = normalize(extractMat3f(getInput().octreeMatrices.octreeToWorld) * get(ray.dir));

    // No hit => sky.

    if (castRes.t > 1.0f)
    {
        float3 c;
        if (I.y >= 0.f)
        {
            float3 horz = { 179.0f, 205.0f, 253.0f };
            float3 zen  = { 77.0f,  102.0f, 179.0f };
            c = horz + (zen - horz) * I.y * I.y;
            c *= 2.5f;
        }
        else
        {
            float3 horz = { 192.0f, 154.0f, 102.0f };
            float3 zen  = { 128.0f, 102.0f, 77.0f };
            c = horz - (zen - horz) * I.y;
        }

        c *= fmaxf(L.y, 0.0f);
        float IL = dot(I, L);
        if (IL > 0.0f) 
            c += make_float3(255.0f, 179.0f, 102.0f) * powf(IL, 1000.0f); // sun

        return toABGR(make_float4(c.x, c.y, c.z, 0.0f));
    }

    // Get voxel color, normal, and ambient.

    float4 voxelColor;
    float3 voxelNormal;
    lookupVoxelColorNormal(voxelColor, voxelNormal, castRes, stack);

    F32 voxelAmbient = 1.0f;
#ifdef VOXELATTRIB_AO
    lookupVoxelAO(voxelAmbient, castRes, stack);
#endif

    // Calculate world-space normal and reflection vectors.

    float3 N  = normalize(getInput().octreeMatrices.octreeToWorldN * voxelNormal);
    float3 R  = (I - N * (dot(N, I) * 2.0f));
    F32    LN = dot(L, N);

    // Cast shadow ray.

    bool shadow = (LN <= 0.0f);
#ifdef ENABLE_SHADOWS
    if (!shadow)
    {
        Ray rayShad;
        rayShad.orig_sz = 0.0f;
        rayShad.dir_sz  = 0.0f;
        rayShad.orig    = castRes.pos + L * 0.0006f;
        rayShad.dir     = L * 3.0f;

        CastResult castResShad;
        CastStack  stackShad;
        castRay(castResShad, stackShad, rayShad);
        shadow = (castResShad.t <= 1.0f);
	}
#endif

    // Shade.

    float4 shadedColor = voxelColor * (voxelAmbient * (0.25f + LN * ((LN < 0.0f) ? 0.15f : (shadow) ? 0.25f : 1.0f)));
    if (!shadow)
        shadedColor += make_float4(32.f, 32.f, 32.f, 0.0f) * powf(fmaxf(dot(L, R), 0.0f), 18.0f); // specular
    shadedColor *= getInput().brightness;

    U32 color = toABGR(shadedColor);

    // Determine post-process filter radius.

    float vSize = (F32)(1 << castRes.stackPtr) / (F32)(1 << CAST_STACK_DEPTH);
    float pSize = ray.orig_sz + castRes.t * ray.dir_sz;
#ifdef JITTER_LOD
    vSize *= vSizeMultiplier;
#endif
    float blurRadius = ::max(vSize / pSize * getInput().maxVoxelSize, 1.0f);

    // Encode in the alpha channel.

    shadedColor.w = log2f(blurRadius) * 32.0f + 0.5f;
    return toABGR(shadedColor);
}

//------------------------------------------------------------------------
// Persistent threads.
//------------------------------------------------------------------------

__device__ void fetchWorkFirst(int& warp, int& batchCounter, int* warpCounter, int batchSize, volatile S32& sharedTemp)
{
#ifdef PERSISTENT_THREADS
    if (threadIdx.x == 0)
        sharedTemp = atomicAdd(warpCounter, batchSize);
    warp = sharedTemp;
    batchCounter = batchSize;
#else
    warp = threadIdx.y + blockIdx.x * RCK_TRACE_BLOCK_HEIGHT;
    batchCounter = 0;
#endif
}

__device__ void fetchWorkNext(int& warp, int& batchCounter, int* warpCounter, int batchSize, volatile S32& sharedTemp)
{
#ifdef PERSISTENT_THREADS
    batchCounter--;
    if (batchCounter > 0)
        warp++;
    else
    {
        if (threadIdx.x == 0)
            sharedTemp = atomicAdd(warpCounter, batchSize);
        batchCounter = batchSize;
        warp = sharedTemp;
    }
#else
    warp = 0x03FFFFFF;
#endif
}

//------------------------------------------------------------------------
// Rendering kernel.
//------------------------------------------------------------------------

#ifdef KERNEL_RENDER

extern "C" __global__ void kernel(void)
{
    const RenderInput& input = getInput();
    __shared__ Aux auxbuf[RCK_TRACE_BLOCK_WIDTH * RCK_TRACE_BLOCK_HEIGHT];
    volatile Aux& aux0 = auxbuf[RCK_TRACE_BLOCK_WIDTH * threadIdx.y];
    volatile Aux& aux  = auxbuf[threadIdx.x + RCK_TRACE_BLOCK_WIDTH * threadIdx.y];

    // fetch first warp of work
    int warp, batchCounter;
    fetchWorkFirst(warp, batchCounter, &g_warpCounter, input.batchSize, aux0.fetchWorkTemp);
    if (warp * 32 >= input.totalWork)
        return; // terminate before starting at all

#ifdef PERSISTENT_THREADS
    // notice that work is being done in this warp slot
    ((S32*)input.activeWarps)[threadIdx.y + blockIdx.x * RCK_TRACE_BLOCK_HEIGHT] = 1;
#endif

    // main warp loop
    for (;;)
    {
        // ray index
        int ridx = warp * 32 + threadIdx.x;
        if (ridx >= input.totalWork)
            return; // terminate individual rays

        // calculate pixel index, position, and frame buffer pointer
        int pidx = (ridx / input.aaRays) % input.numPrimaryRays;
        int ppos;

        if (input.flags & RenderFlags_CoarsePass)
        {
            ppos = tex1Dfetch(texIndexToPixelCoarse, pidx);
            aux.framePtr = (U32*)input.frameCoarse + ppos;
        }
        else
        {
            ppos = tex1Dfetch(texIndexToPixel, pidx);
            aux.framePtr = (U32*)input.frame + ppos;
#ifdef LARGE_RECONSTRUCTION_KERNEL
            aux.aaSamplePtr = (U32*)input.aaSampleBuffer + ppos * input.aaRays + (ridx % input.aaRays);
#endif
        }

        // construct ray
        Ray ray = constructPrimaryRay(ppos, ridx, aux);
        aux.ray.orig.x = ray.orig.x;
        aux.ray.orig.y = ray.orig.y;
        aux.ray.orig.z = ray.orig.z;
        aux.ray.dir.x = ray.dir.x;
        aux.ray.dir.y = ray.dir.y;
        aux.ray.dir.z = ray.dir.z;
        aux.ray.orig_sz = ray.orig_sz;
        aux.ray.dir_sz = ray.dir_sz;

        if (getInput().flags & RenderFlags_CoarsePass)
        {
            CastResult castRes;
            CastStack stack;
            castRay(castRes, stack, aux.ray);
            if (castRes.t < 1.0f)
            {
                F32 size = (F32)(1 << castRes.stackPtr) / (F32)(1 << CAST_STACK_DEPTH);
                castRes.t -= size / length(get(aux.ray.dir)) * 0.5f;
            }
            *(float*)aux.framePtr = ::max(castRes.t, 0.0f);
        } else
        {
#ifdef JITTER_LOD
            U32 color = processPrimaryRay(aux.ray, aux.vSizeMultiplier);
#else
            U32 color = processPrimaryRay(aux.ray, aux.ray.orig.x);
#endif

            // write results
            if (input.aaRays == 1)
                *aux.framePtr = color; // no AA
            else
            {
#ifdef LARGE_RECONSTRUCTION_KERNEL
                *aux.aaSamplePtr = color; // individual sample result
#endif
                // unpack result
                U32 resc = (color & 0xff) | ((color & 0xff00) << 2) | ((color & 0xff0000) << 4);
                aux.aa.color = resc;  // rgb with bits shifted up
                aux.aa.alpha = color; // original color

                // sum with one thread
                if ((threadIdx.x & 3) == 0)
                {
                    // rgb
                    U32 resc0 = (&aux)[0].aa.color;
                    U32 resc1 = (&aux)[1].aa.color;
                    U32 resc2 = (&aux)[2].aa.color;
                    U32 resc3 = (&aux)[3].aa.color;
                    resc = (resc0 + resc1 + resc2 + resc3);
                    resc = ((resc >> 2) & 0xff) | ((resc >> 4) & 0xff00) | ((resc >> 6) & 0xff0000);

                    // alpha
                    U32 resa0 = (&aux)[0].aa.alpha;
                    U32 resa1 = (&aux)[1].aa.alpha;
                    U32 resa2 = (&aux)[2].aa.alpha;
                    U32 resa3 = (&aux)[3].aa.alpha;
                    U32 resa = ::min(::min(resa0, resa1), ::min(resa2, resa3));;

                    // combine min alpha and avg color
                    *aux.framePtr = (resa & 0xff000000) | resc;
                }
            }
        }

        // fetch more work
        fetchWorkNext(warp, batchCounter, &g_warpCounter, input.batchSize, aux0.fetchWorkTemp);
    }
}

#endif

//------------------------------------------------------------------------
// Performance measurement kernel.
//------------------------------------------------------------------------

#ifdef KERNEL_RAYCAST_PERF

extern "C" __global__ void kernel(void)
{
    const RenderInput& input = getInput();
    __shared__ Aux auxbuf[RCK_TRACE_BLOCK_WIDTH * RCK_TRACE_BLOCK_HEIGHT];
    volatile Aux& aux0 = auxbuf[RCK_TRACE_BLOCK_WIDTH * threadIdx.y];
    volatile Aux& aux  = auxbuf[threadIdx.x + RCK_TRACE_BLOCK_WIDTH * threadIdx.y];

    // fetch first warp of work
    int warp, batchCounter;
    fetchWorkFirst(warp, batchCounter, &g_warpCounter, input.batchSize, aux0.fetchWorkTemp);
    if (warp * 32 >= input.totalWork)
        return; // terminate before starting at all

#ifdef PERSISTENT_THREADS
    // notice that work is being done in this warp slot
    ((S32*)input.activeWarps)[threadIdx.y + blockIdx.x * RCK_TRACE_BLOCK_HEIGHT] = 1;
#endif

    // main warp loop
    for (;;)
    {
        // ray index
        int ridx = warp * 32 + threadIdx.x;
        if (ridx >= input.totalWork)
            return; // terminate individual rays

        // calculate pixel index, position, and frame buffer pointer
        int pidx = ridx % input.numPrimaryRays;
        int ppos;

        if (input.flags & RenderFlags_CoarsePass)
        {
            ppos = tex1Dfetch(texIndexToPixelCoarse, pidx);
            aux.framePtr = (U32*)input.frameCoarse + ppos;
        }
        else
        {
            ppos = tex1Dfetch(texIndexToPixel, pidx);
            aux.framePtr = (U32*)input.frame + ppos;
        }

        // construct ray
        Ray ray = constructPrimaryRay(ppos, pidx, aux);
        aux.ray.orig.x = ray.orig.x;
        aux.ray.orig.y = ray.orig.y;
        aux.ray.orig.z = ray.orig.z;
        aux.ray.dir.x = ray.dir.x;
        aux.ray.dir.y = ray.dir.y;
        aux.ray.dir.z = ray.dir.z;
        aux.ray.orig_sz = ray.orig_sz;
        aux.ray.dir_sz = ray.dir_sz;

        CastResult castRes;
        CastStack stack;
        castRay(castRes, stack, aux.ray);
        if (castRes.t < 1.0f)
        {
            F32 size = (F32)(1 << castRes.stackPtr) / (F32)(1 << CAST_STACK_DEPTH);
            castRes.t -= size / length(get(aux.ray.dir)) * 0.5f;
        }
        *(float*)aux.framePtr = castRes.t;

        // fetch more work
        fetchWorkNext(warp, batchCounter, &g_warpCounter, input.batchSize, aux0.fetchWorkTemp);
    }
}

#endif

//------------------------------------------------------------------------
// Post-process filter kernel.
//------------------------------------------------------------------------

extern "C" __global__ void blurKernel(void)
{
    const RenderInput& input = getInput();
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int cx = input.frameSize.x;
    int cy = input.frameSize.y;

    if (px >= cx || py >= cy)
        return;

    U32* pResult = ((U32*)input.frame) + (px + cx*py);

    float4 ccol = tex1Dfetch(texTempFrameIn, px + cx*py);
    float rad = exp2f(ccol.w * (255.f / 32.f)) * BLUR_FACTOR;

    if (rad <= 1.f)
    {
        // single-pixel case
#ifdef LARGE_RECONSTRUCTION_KERNEL
        if (input.aaRays == 4)
        {
            int apos = (px + cx*py) * 4;
            cx *= 4;

            ccol *= 4.f;
            ccol += tex1Dfetch(texAASamplesIn, apos-4 +2);
            ccol += tex1Dfetch(texAASamplesIn, apos-4 +3);
            ccol += tex1Dfetch(texAASamplesIn, apos+4 +0);
            ccol += tex1Dfetch(texAASamplesIn, apos+4 +1);
            ccol += tex1Dfetch(texAASamplesIn, apos-cx+1);
            ccol += tex1Dfetch(texAASamplesIn, apos-cx+2);
            ccol += tex1Dfetch(texAASamplesIn, apos+cx+0);
            ccol += tex1Dfetch(texAASamplesIn, apos+cx+3);
            ccol *= (1.f/12.f);
            ccol.w = 1.f;
            *pResult = toABGR(ccol * 255.f);
        } else
#endif
        {
            ccol.w = 1.f;
            *pResult = toABGR(ccol * 255.f);
        }
        return;
    }

    float4 accum = {0, 0, 0, 0};
    for (int i=0; i < BLUR_LUT_SIZE; i++)
    {
        int4 b = c_blurLUT[i];
        float d = __int_as_float(b.w);
        if (d >= rad)
            break;

        int x = px + b.x;
        int y = py + b.y;

        float w = __int_as_float(b.z);

        if (x < 0) w = 0.f;
        if (y < 0) w = 0.f;
        if (x >= input.frameSize.x) w = 0.f;
        if (y >= input.frameSize.y) w = 0.f;

        float4 c = tex1Dfetch(texTempFrameIn, x + __mul24(cx, y));
        float rad2 = exp2f(c.w * (255.f / 32.f)) * BLUR_FACTOR;
        if (w > 0.f)
            rad = ::min(rad, rad2);

        w *= fminf(fmaxf(rad - d, 0.f), 1.f);

        accum.x += c.x * w;
        accum.y += c.y * w;
        accum.z += c.z * w;
        accum.w += w;
    }

    float invw = 1.f / accum.w;
    accum.x *= invw;
    accum.y *= invw;
    accum.z *= invw;

    accum.w = 1.f;
    *pResult = toABGR(accum * 255.f);
}

//------------------------------------------------------------------------
