#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2011, NVIDIA Corporation
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *      * Redistributions of source code must retain the above copyright
 *        notice, this list of conditions and the following disclaimer.
 *      * Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimer in the
 *        documentation and/or other materials provided with the distribution.
 *      * Neither the name of NVIDIA Corporation nor the
 *        names of its contributors may be used to endorse or promote products
 *        derived from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 *  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 *  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "Ambient.hpp"
#include "../io/OctreeRuntime.hpp"
#include "../io/AttachIO.hpp"

using namespace FW;

//------------------------------------------------------------------------
// Global variables.
//------------------------------------------------------------------------

__constant__ int4 c_input[(sizeof(AmbientInput) + sizeof(int4) - 1) / sizeof(int4)];
__device__ S32 g_warpCounter;

//------------------------------------------------------------------------
// Common helper functions.
//------------------------------------------------------------------------

__device__ inline const AmbientInput& getInput(void)
{
    return *(const AmbientInput*)c_input;
}

__device__ inline void updateCounter(PerfCounter counter, int amount = 1)
{
}

__device__ inline void updateCountersForGlobalAccess (int sizeLog2, S32* addr)
{
}

__device__ inline void updateCountersForLocalAccess  (int sizeLog2, int id)
{
}

//------------------------------------------------------------------------
// Utility routines.
//------------------------------------------------------------------------

#include "Util.inl"
#include "Raycast.inl"
#include "AttribLookup.inl"

//------------------------------------------------------------------------
// Private definitions.
//------------------------------------------------------------------------

#define F3COPY(a,b) a.x=b.x, a.y=b.y, a.z=b.z

struct Aux
{
    float  rx;
    float  ry;
    float3 normal;
    float3 orig;
    float  pad;
};

// 2 times S16 packed into a 32-bit value
__constant__ S32 c_aotable[256] = {
    0x9029cc81,0x92f8c801,0xa31fb481,0xa5eeb001,0xac40c201,0xae5ca701,0xb4aecb01,
    0xb6c9b901,0xbb009501,0xbc68a881,0xbf37bc01,0xc2bac081,0xc7a4b301,0xc90c9c81,
    0xcbdb9801,0xd012a101,0xd4488f01,0xd5b0af01,0xd664ca01,0xd87fb801,0xdcb69101,
    0xe0edb501,0xe2549881,0xe73fc701,0xe95aa601,0xed918b01,0xeef9bc81,0xf1c8a001,
    0xf54bc881,0xf5fe8e01,0xfa35bb01,0xfb9d8081,0x8384f081,0x8654ec01,0x8a8ada01,
    0x8ec1e301,0x972ffe01,0x9b65d701,0x9ccdef01,0x9f9cf801,0xa3d3d101,0xa80af501,
    0xa971d881,0xb077e601,0xb615fc81,0xb8e5e001,0xbd1bce01,0xc152fb01,0xc589d401,
    0xc9c0e901,0xcdf6dd01,0xcf5ee481,0xd22df201,0xd5560000,0xda9bfd01,0xdc02d481,
    0xded1d001,0xe308ee01,0xe8a6e081,0xeb76f401,0xefacd901,0xf3e3e501,0xf81adc01,
    0xfc51fa01,0xfe6c9401,0x02a2a901,0x06d89d01,0x0840a481,0x08f4c101,0x0b0fb201,
    0x0f469a01,0x1161c401,0x137da301,0x14e48c81,0x17b38801,0x1beabe01,0x20219701,
    0x2189b081,0x2458ac01,0x27dbc281,0x2aaac001,0x2cc5a501,0x30fc9c01,0x3533ba01,
    0x39699301,0x3ad1ac81,0x3b85c901,0x3da0a801,0x4123c481,0x460eb701,0x47759f01,
    0x4c60c301,0x4e7bb101,0x541ab881,0x5f56ab01,0x6e16c601,0x0086d301,0x01eeec81,
    0x04bde801,0x0d2bf701,0x0e92df01,0x1598f101,0x19cfcd01,0x1b36f881,0x1e06e201,
    0x223cd601,0x2673eb01,0x2ee1f601,0x3317db01,0x347fff01,0x374ee401,0x3fbced01,
    0x43f2d201,0x4829ea01,0x4dc7f481,0x5097f001,0x54cdde01,0x5904e701,0x5a6cd081,
    0x5d3bcc01,0x6172f901,0x65a8d501,0x6710e881,0x69dffc01,0x724df301,0x73b4dc81,
    0x7683d801,0x7abae101,0x810f09ff,0x86ae147f,0x897d0fff,0x8db32dff,0x91ea06ff,
    0x9352207f,0x9a5818ff,0x9e8e24ff,0x9ff6087f,0xa2c51bff,0xab3312ff,0xac9a2c7f,
    0xaf6927ff,0xb3a000ff,0xb93f1eff,0xbc0e03ff,0xc04430ff,0xc47b0cff,0xc8b221ff,
    0xcce915ff,0xd11f2aff,0xd287027f,0xd6be227f,0xddc41aff,0xe1fa23ff,0xe63108ff,
    0xea682cff,0xebcf047f,0xee9f11ff,0xf2d529ff,0xf70c02ff,0xfb432fff,0x962133ff,
    0x9c733fff,0xa6fc39ff,0xad4e48ff,0xb2ec447f,0xb5bc51ff,0xb7d736ff,0xb9f269ff,
    0xbe2942ff,0xc2606fff,0xc5e3387f,0xc6975dff,0xcacd66ff,0xcc35507f,0xcf044bff,
    0xd77244ff,0xd8d9707f,0xd98d35ff,0xdba86bff,0xdf2b3eff,0xdfdf59ff,0xe41662ff,
    0xe57d4c7f,0xe84d47ff,0xec837dff,0xf0ba56ff,0xf2226eff,0xf4f177ff,0xf874347f,
    0xf92850ff,0xfd5e74ff,0xfec6587f,0xff7a1dff,0x03af26ff,0x0517107f,0x07e60bff,
    0x105414ff,0x11bb287f,0x18c105ff,0x1cf832ff,0x1e5f1c7f,0x212f17ff,0x256520ff,
    0x299c0eff,0x2b042eff,0x320a10ff,0x37a8187f,0x3a7701ff,0x3eae25ff,0x42e50aff,
    0x471b1fff,0x4b520dff,0x50f0007f,0x53c013ff,0x57f628ff,0x5c2d1cff,0x5d95247f,
    0x606431ff,0x649b19ff,0x68d122ff,0x6a390c7f,0x6d0807ff,0x757616ff,0x7de304ff,
    0x019441ff,0x05cb65ff,0x0a024aff,0x0b697c7f,0x0c1d38ff,0x0e385fff,0x126f4dff,
    0x148a3bff,0x16a67aff,0x180d407f,0x1add53ff,0x1f1368ff,0x234a5cff,0x24b2647f,
    0x278171ff,0x2bb849ff,0x2dd337ff,0x3156547f,0x34254fff,0x364034ff,0x385c6dff,
    0x3c9346ff,0x3dfa607f,0x444c3c7f,0x450058ff,0x493764ff,0x4a9e487f,0x4d6e5bff,
    0x4f893aff,0x55db52ff,0x5e4940ff,0x66b643ff
};

//------------------------------------------------------------------------

extern "C" __global__ void ambientKernel(void)
{
    const AmbientInput& input = getInput();
    __shared__ Aux auxbuf[AMBK_BLOCK_WIDTH * AMBK_BLOCK_HEIGHT];
    volatile S32& aux0 = *((S32*)&auxbuf[AMBK_BLOCK_WIDTH * threadIdx.y]);
    volatile Aux& aux  = auxbuf[threadIdx.x + AMBK_BLOCK_WIDTH * threadIdx.y];

    const OctreeMatrices& mtx = getInput().octreeMatrices;

    // fetch first warp of work
    if (threadIdx.x == 0)
        aux0 = atomicAdd(&g_warpCounter, 1);
    int warp = aux0;
    if (warp >= input.numRequests)
        return; // terminate before starting at all

    // notice that work is being done in this warp slot
    ((S32*)input.activeWarps)[threadIdx.y + blockIdx.x * AMBK_BLOCK_HEIGHT] = 1;

    CastResult castRes;
    CastStack stack;

    // main warp loop
    for (;;)
    {
        // request index
        int ridx = warp;
        if (ridx >= input.numRequests)
            return;

        {
            AmbientRequest& req = ((AmbientRequest*)input.requestPtr)[ridx];

            // construct node position
            const U64* node     = (const U64*)getInput().rootNode;
            S32        stackPtr = CAST_STACK_DEPTH - 1;
            int        rlevel   = req.level;
            int        cidx     = 0;

            // find the node
            do
            {
                // determine child idx
                U32 smask = 1 << stackPtr;
                cidx = 0;
                if (req.pos.x & smask) cidx |= 1;
                if (req.pos.y & smask) cidx |= 2;
                if (req.pos.z & smask) cidx |= 4;

                if (stackPtr <= rlevel)
                    break;

                // move down
                U32 nodeData = *(const U32*)node;
                S32 bits = nodeData << (8-cidx);
                stack.write(stackPtr, (S32*)node, 0.0f);
                stackPtr--;
                int ofs = nodeData >> 17;
                node += (nodeData & 0x10000) ? *(const S32*)(node + ofs) : ofs;
                node += popc8(bits & 0xFF);
            }
            while (stackPtr >= 0); // always true

            // construct request position in float
            float3 rpos;
            rpos.x  = __int_as_float(req.pos.x + 0x3f800000u);
            rpos.y  = __int_as_float(req.pos.y + 0x3f800000u);
            rpos.z  = __int_as_float(req.pos.z + 0x3f800000u);

            // set up position struct
            castRes.node     = (S32*)node;
            castRes.stackPtr = stackPtr;
            castRes.childIdx = cidx;
            castRes.pos      = rpos;

            float3 orig = rpos;

            // sample color and normal at request position, adjust ray origin
            F32 vsize = __int_as_float((127 - ::min(CAST_STACK_DEPTH - rlevel, 13)) << 23);
            float4 color; // dummy
            float3 normal;
            lookupVoxelColorNormal(color, normal, castRes, stack);
                normal = normalize(normal);
                float nlen = 1.f / fmaxf3(fabsf(normal.x), fabsf(normal.y), fabsf(normal.z));
                orig += normal * (vsize * nlen);

            F3COPY(aux.normal, normal);
            F3COPY(aux.orig, orig);

            // construct 2d rotation for samples
            U32 ix = __float_as_int(rpos.x);
            U32 iy = __float_as_int(rpos.y);
            U32 iz = __float_as_int(rpos.z);
            jenkinsMix(ix, iy, iz);
            ix ^= req.level;
            float rx, ry, rlen;
            do
            {
                jenkinsMix(ix, iy, iz);
                rx = (float)ix / (4.f * (1u << 30)) * 2.f - 1.f;
                ry = (float)iy / (4.f * (1u << 30)) * 2.f - 1.f;
                rlen = rx*rx+ry*ry;
            } while (rlen > 1.f);
            rlen = rsqrtf(rlen);
            aux.rx = rx * rlen;
            aux.ry = ry * rlen;
        }

        // construct ray
        Ray ray;
        F3COPY(ray.orig, aux.orig);
        ray.orig_sz = 0.f;
        ray.dir_sz  = 0.f;

        // light vector
        float3 L = { -.4f, .5f, -.3f };
        L = normalize(L);

        // cast the ao rays
        float3 illum;
#ifdef FLIP_NORMALS
        for (int pass = 0; pass < 2; pass++)
#endif
        {
            illum = make_float3(0.f, 0.f, 0.f);
            for (int i=threadIdx.x; i < input.raysPerNode; i += 32)
            {
                // use ao table
                S32 ao32 = c_aotable[i];
                float sy = (float)ao32 * __int_as_float(0x30000000);
                ao32 <<= 16;
                float sx = (float)ao32 * __int_as_float(0x30000000);

                // rotate in 2d
                float x = aux.rx*sx + aux.ry*sy;
                float y = aux.ry*sx - aux.rx*sy;

                // construct basis for normal
                float3 normal;
                F3COPY(normal, aux.normal);
                float3 b1 = normalize(perpendicular(normal));
                float3 b2 = cross(normal, b1);

                // set ray direction
                float z = sqrtf(fabsf(1.f - x*x - y*y));
                ray.dir = x*b1 + y*b2 + z*normal;
                ray.dir *= input.rayLength;

#ifdef FLIP_NORMALS
                if (pass == 1)
                    ray.dir *= -1.0f;
#endif

                // cast the ray
                CastResult castResRay;
                CastStack  stackRay;
                castRay(castResRay, stackRay, ray);

                float ill = smoothstep(castResRay.t * 2.f - 1.f); // taper off in last 50%
                illum.x += ill;
                illum.y += ill;
                illum.z += ill;
            }

            // calculate result
            illum *= (1.f / input.raysPerNode);

            // sum over warp
            F3COPY(aux.orig, illum);
            if (!(threadIdx.x & 1))  aux.orig.x+=(&aux+ 1)->orig.x,aux.orig.y+=(&aux+ 1)->orig.y,aux.orig.z+=(&aux+ 1)->orig.z;
            if (!(threadIdx.x & 2))  aux.orig.x+=(&aux+ 2)->orig.x,aux.orig.y+=(&aux+ 2)->orig.y,aux.orig.z+=(&aux+ 2)->orig.z;
            if (!(threadIdx.x & 4))  aux.orig.x+=(&aux+ 4)->orig.x,aux.orig.y+=(&aux+ 4)->orig.y,aux.orig.z+=(&aux+ 4)->orig.z;
            if (!(threadIdx.x & 8))  aux.orig.x+=(&aux+ 8)->orig.x,aux.orig.y+=(&aux+ 8)->orig.y,aux.orig.z+=(&aux+ 8)->orig.z;
            if (!(threadIdx.x & 16)) aux.orig.x+=(&aux+16)->orig.x,aux.orig.y+=(&aux+16)->orig.y,aux.orig.z+=(&aux+16)->orig.z;

#ifdef FLIP_NORMALS
            if (auxbuf[AMBK_BLOCK_WIDTH * threadIdx.y].orig.x >= 0.1f)
                break;

            AmbientRequest& req = ((AmbientRequest*)input.requestPtr)[ridx];
            ray.orig.x = __int_as_float(req.pos.x + 0x3f800000u) * 2.0f - ray.orig.x;
            ray.orig.y = __int_as_float(req.pos.y + 0x3f800000u) * 2.0f - ray.orig.y;
            ray.orig.z = __int_as_float(req.pos.z + 0x3f800000u) * 2.0f - ray.orig.z;
#endif
        }

        // write result
        if (threadIdx.x == 0)
        {
            AmbientResult& res = ((AmbientResult*)input.resultPtr)[ridx];
            F3COPY(res.ao, aux.orig);
        }

        // fetch more work
        if (threadIdx.x == 0)
            aux0 = atomicAdd(&g_warpCounter, 1);
        warp = aux0;
    }
}

//------------------------------------------------------------------------
